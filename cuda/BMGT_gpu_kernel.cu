#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>

typedef int bin_enc;

template <int _curr_subjs, int _variants, int prior_numer>
__global__ void set_prior_probs(float *_post_probs)
{
    const float pi0 = (float)(prior_numer) / 100.0;
    int s_iter = blockIdx.x * blockDim.x + threadIdx.x;
    float prob = 1.0;
    for (int i = 0; i < _curr_subjs * _variants; i++)
    {
        if ((s_iter & (1 << i)) == 0)
            prob *= pi0;
        else
            prob *= (1.0 - pi0);
    }
    _post_probs[s_iter] = prob;
}

template <int _curr_subjs, int _variants>
__global__ void halving_serial_kernel_V1(const float *_post_probs, float *partition_mass, bin_enc *candidate)
{
    int partition_id = 0;
    int s_iter = blockIdx.x * blockDim.x + threadIdx.x;
    for (bin_enc experiment = 0; experiment < (1 << _curr_subjs); experiment++)
    {
        // #pragma unroll
        // for (int variant = 0; variant < _variants; variant++)
        // {
        //     partition_id |= ((1 << variant) & (((experiment & (s_iter >> (variant * _curr_subjs))) - experiment) >> 31));
        // }

        partition_id |= (1 & (((experiment & s_iter) - experiment) >> 31));
        partition_id |= (2 & (((experiment & (s_iter >> _curr_subjs)) - experiment) >> 31));
        atomicAdd(&partition_mass[experiment * (1 << _variants) + partition_id], _post_probs[s_iter]);
        // partition_mass[experiment * (1 << _variants) + partition_id] += _post_probs[s_iter];
        partition_id = 0;
    }

    // __syncthreads();

    // float temp = 0.0;
    // float prob = 1.0 / (1 << _variants);
    // int experiment = blockIdx.x * blockDim.x + threadIdx.x;
    // if (experiment < (1 << _curr_subjs))
    // {
    //     for (bin_enc i = 0; i < (1 << _variants); i++)
    //     {
    //         temp += abs(partition_mass[experiment * (1 << _variants) + i] - prob);
    //     }
    //     if(temp < atomicMinFloat(&global_min, temp)){
    //         atomicCAS(&global_candidate, global_candidate, experiment);
    //     }

    // }
}

template <int _curr_subjs, int _variants>
__global__ void halving_serial_kernel(const float *_post_probs, float *partition_mass, bin_enc *candidate)
{
    volatile int partition_id = 0;
    __shared__ float block_partition_mass[(1 << _curr_subjs) * (1 << _variants)];
    int s_iter = blockIdx.x * blockDim.x + threadIdx.x;
    for (bin_enc experiment = threadIdx.x; experiment < (1 << _curr_subjs) + threadIdx.x; experiment++)
    {
        // #pragma unroll
        // for (int variant = 0; variant < _variants; variant++)
        // {
        //     partition_id |= ((1 << variant) & (((experiment & (s_iter >> (variant * _curr_subjs))) - experiment) >> 31));
        // }
        int ex = experiment % blockDim.x;

        partition_id |= (1 & (((ex & s_iter) - ex) >> 31));
        partition_id |= (2 & (((ex & (s_iter >> _curr_subjs)) - ex) >> 31));
        block_partition_mass[ex * (1 << _variants) + partition_id] += _post_probs[s_iter];
        partition_id = 0;
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {
        for (int i = 0; i < (1 << _curr_subjs) * (1 << _variants); i++)
        {
            atomicAdd(&partition_mass[i], block_partition_mass[i]);
        }
    }
    __syncthreads();

    // __syncthreads();

    // float temp = 0.0;
    // float prob = 1.0 / (1 << _variants);
    // int experiment = blockIdx.x * blockDim.x + threadIdx.x;
    // if (experiment < (1 << _curr_subjs))
    // {
    //     for (bin_enc i = 0; i < (1 << _variants); i++)
    //     {
    //         temp += abs(partition_mass[experiment * (1 << _variants) + i] - prob);
    //     }
    //     if(temp < atomicMinFloat(&global_min, temp)){
    //         atomicCAS(&global_candidate, global_candidate, experiment);
    //     }

    // }
}

int main()
{
    float *post_probs, *partition_mass;
    constexpr int curr_subjs = 12;
    constexpr int variants = 2;
    constexpr int prior_numer = 1;

    std::cout << "N = " << curr_subjs << ", k = " << variants << ", prior = " << (float)(prior_numer) / 100.0 << std::endl;
    int numElements = (1 << (curr_subjs * variants));
    hipError_t cudaStatus = hipMalloc((void **)&post_probs, numElements * sizeof(float));
    cudaStatus = hipMemset(post_probs, 1.0 / numElements, numElements * sizeof(float));

    cudaStatus = hipMalloc((void **)&partition_mass, (1 << curr_subjs) * (1 << variants) * sizeof(float));
    cudaStatus = hipMemset(partition_mass, 0.0, (1 << curr_subjs) * (1 << variants) * sizeof(float));

    bin_enc *d_candidate;
    hipMalloc((void **)&d_candidate, sizeof(bin_enc));

    dim3 blockDims(256);                                          // Adjust block dimensions as needed
    dim3 gridDims((numElements + blockDims.x - 1) / blockDims.x); // Calculate grid dimensions

    std::chrono::time_point<std::chrono::system_clock> start, end;
    start = std::chrono::system_clock::now();
    set_prior_probs<curr_subjs, variants, prior_numer><<<gridDims, blockDims>>>(post_probs);

    end = std::chrono::system_clock::now();
    std::chrono::duration<float> elapsedSeconds = end - start;

    std::cout << "Prior kernel execution time: " << elapsedSeconds.count() << " seconds" << std::endl;

    start = std::chrono::system_clock::now();

    halving_serial_kernel_V1<curr_subjs, variants><<<gridDims, blockDims>>>(post_probs, partition_mass, d_candidate);

    hipDeviceSynchronize(); // Wait for the kernel to finish

    end = std::chrono::system_clock::now();
    elapsedSeconds = end - start;

    std::cout << "BBPA kernel execution time: " << elapsedSeconds.count() << " seconds" << std::endl;

    // Copy the result back from the GPU
    bin_enc h_candidate;
    hipMemcpy(&h_candidate, d_candidate, sizeof(bin_enc), hipMemcpyDeviceToHost);

    float *h_partition_mass = new float[(1 << curr_subjs) * (1 << variants)];
    hipMemcpy(h_partition_mass, partition_mass, (1 << curr_subjs) * (1 << variants) * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < (1 << curr_subjs) * (1 << variants); i+=4)
    {
        std::cout << h_partition_mass[i] + h_partition_mass[i+1] + h_partition_mass[i+2] + h_partition_mass[i+3] << "  ";
    }

    std::cout << "\nCandidate is: " << h_candidate << std::endl;

    // Free allocated memory on the GPU
    hipFree(post_probs);
    hipFree(partition_mass);
    hipFree(d_candidate);

    return 0;
}
