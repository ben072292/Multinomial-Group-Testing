#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>

#ifndef NUM
#define NUM 13
#endif

#ifndef K
#define K 2
#endif

#ifndef SMEM
#define SMEM 9
#endif

#ifndef BLOCK
#define BLOCK 256
#endif

typedef int bin_enc;

template <int N, int k, int prior_numer>
__global__ void set_prior_probs(float *_post_probs)
{
    const float pi0 = (float)(prior_numer) / 100.0;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float prob = 1.0f;
    for (int i = 0; i < N * k; i++)
    {
        if ((tid & (1 << i)) == 0)
            prob *= pi0;
        else
            prob *= (1.0f - pi0);
    }
    _post_probs[tid] = prob;
}

/** RTX3060: N = 11, k = 2, prior = 0.3, block 256: 0.198432 seconds
 *  A100: N = 15, k = 2, prior = 0.1, block 1024: 30.4172 seconds
*/
template <int N, int k>
__global__ void BBPA_target(const float *__restrict__ _post_probs, float *__restrict__ partition_mass)
{
    int partition_id = 0;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float val = _post_probs[tid];
    for (bin_enc ex = 0; ex < (1 << N); ex++)
    {
        // #pragma unroll
        // for (int variant = 0; variant < k; variant++)
        // {
        //     partition_id |= ((1 << variant) & (((experiment & (tid >> (variant * N))) - experiment) >> 31));
        // }

        partition_id |= (1 & (((ex & tid) - ex) >> 31));
        partition_id |= (2 & (((ex & (tid >> N)) - ex) >> 31));
        // atomicAdd(&partition_mass[ex * (1 << k) + partition_id], val);
        partition_mass[ex * (1 << k) + partition_id] = val;
        partition_id = 0;
    }
}

/** RTX3060: N = 11, k = 2, prior = 0.3 block 256: 0.844862 seconds
 *  A100: N = 15, k = 2, prior = 0.1 blick 256: 736.655 seconds
*/
template <int N, int k>
__global__ void BBPA_write_aligned(const float *__restrict__ _post_probs, float *__restrict__ partition_mass)
{
    int partition_id = 0;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float val = _post_probs[tid];
    for (bin_enc ex = 0; ex < (1 << N); ex++)
    {
        // #pragma unroll
        // for (int variant = 0; variant < k; variant++)
        // {
        //     partition_id |= ((1 << variant) & (((experiment & (tid >> (variant * N))) - experiment) >> 31));
        // }

        partition_id |= (1 & (((ex & tid) - ex) >> 31));
        partition_id |= (2 & (((ex & (tid >> N)) - ex) >> 31));
        atomicAdd(&partition_mass[ex * (1 << k) + partition_id], val);
        // partition_mass[ex * (1 << k) + partition_id] += val;
        partition_id = 0;
    }
}

/** N = 11, k = 2, prior = 0.3 block 256: 0.580698 seconds
 *  N = 15, k = 2, prior = 0.1 block 256: 335.417 seconds
*/
template <int N, int k>
__global__ void BBPA(const float *__restrict__ _post_probs, float *__restrict__ partition_mass)
{
    int partition_id = 0;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float val = _post_probs[tid];
    for (bin_enc experiment = threadIdx.x; experiment < (1 << N) + threadIdx.x; experiment++)
    {
        // #pragma unroll
        // for (int variant = 0; variant < k; variant++)
        // {
        //     partition_id |= ((1 << variant) & (((experiment & (tid >> (variant * N))) - experiment) >> 31));
        // }

        int ex = experiment % (1 << N);
        partition_id |= (1 & (((ex & tid) - ex) >> 31));
        partition_id |= (2 & (((ex & (tid >> N)) - ex) >> 31));
        atomicAdd(&partition_mass[ex * (1 << k) + partition_id], val);
        // partition_mass[experiment * (1 << k) + partition_id] += val;
        partition_id = 0;
    }
}

/** N = 11, k = 2, prior = 0.3, block 256: 0.0725678 seconds*/
template <int N, int k>
__global__ void BBPA_smem_interleave(const float *__restrict__ _post_probs, float *__restrict__ partition_mass)
{
    __shared__ float block_partition_mass[(1 << N) * (1 << k)];
    if (threadIdx.x == 0)
        memset(block_partition_mass, 0, (1 << N) * (1 << k) * sizeof(float));

    __syncthreads();
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (bin_enc experiment = threadIdx.x; experiment < (1 << N) + threadIdx.x; experiment++)
    {
        // #pragma unroll
        // for (int variant = 0; variant < k; variant++)
        // {
        //     partition_id |= ((1 << variant) & (((experiment & (tid >> (variant * N))) - experiment) >> 31));
        // }
        int ex = experiment % (1 << N);

        int partition_id = ((1 & (((ex & tid) - ex) >> 31))) | ((2 & (((ex & (tid >> N)) - ex) >> 31)));
        block_partition_mass[ex * (1 << k) + partition_id] += _post_probs[tid];
        __syncthreads();
    }
    // __syncthreads();
    if (threadIdx.x == 0)
    {
        for (int i = 0; i < (1 << N) * (1 << k); i++)
        {
            atomicAdd(&partition_mass[i], block_partition_mass[i]);
        }
    }
}

/** RTX3060: N = 11, k = 2, prior = 0.3, block 256: 0.0725678 seconds
 *  A100: N = 15, k = 2, prior = 0.3, block 256: 82.3447 seconds
 *  A100: N = 15, k = 2, prior = 0.3, block 256: 69.4607 seconds
*/
template <int N, int k, int smem>
__global__ void BBPA_smem_interleave(const float *__restrict__ _post_probs, float *__restrict__ partition_mass)
{
    __shared__ float block_partition_mass[(1 << (smem + k))];
    if (threadIdx.x == 0)
        memset(block_partition_mass, 0, (1 << (smem + k)) * sizeof(float));

    __syncthreads();
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (int iter = 0; iter < (1 << (N - smem)); iter++)
    {
        for (bin_enc experiment = threadIdx.x; experiment < (1 << smem) + threadIdx.x; experiment++)
        {
            // #pragma unroll
            // for (int variant = 0; variant < k; variant++)
            // {
            //     partition_id |= ((1 << variant) & (((experiment & (tid >> (variant * N))) - experiment) >> 31));
            // }
            int ex = (experiment % (1 << smem)) + iter * (1 << smem);

            int partition_id = ((1 & (((ex & tid) - ex) >> 31))) | ((2 & (((ex & (tid >> N)) - ex) >> 31)));
            block_partition_mass[(experiment % (1 << smem)) * (1 << k) + partition_id] += _post_probs[tid];
            __syncthreads();
        }
        // __syncthreads();
        if (threadIdx.x == 0)
        {
            for (int i = 0; i < (1 << (smem + k)); i++)
            {
                atomicAdd(&partition_mass[iter * (1 << (smem + k)) + i], block_partition_mass[i]);
            }
            memset(block_partition_mass, 0, (1 << (smem + k)) * sizeof(float));
        }
        __syncthreads();
    }
}

/** N = 11, k = 2, prior = 0.3, block 256, BBPA kernel execution time: 0.829783 seconds*/
template <int N, int k>
__global__ void BBPA_smem(const float *__restrict__ _post_probs, float *partition_mass)
{
    __shared__ float block_partition_mass[(1 << N) * (1 << k)];
    if (threadIdx.x == 0)
        memset(block_partition_mass, 0, (1 << N) * (1 << k) * sizeof(float));

    __syncthreads();
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (bin_enc ex = 0; ex < (1 << N); ex++)
    {
        // #pragma unroll
        // for (int variant = 0; variant < k; variant++)
        // {
        //     partition_id |= ((1 << variant) & (((experiment & (tid >> (variant * N))) - experiment) >> 31));
        // }

        int partition_id = ((1 & (((ex & tid) - ex) >> 31))) | ((2 & (((ex & (tid >> N)) - ex) >> 31)));
        // __syncthreads();
        atomicAdd(&block_partition_mass[ex * (1 << k) + partition_id], _post_probs[tid]);
        // __syncthreads();
    }
    if (threadIdx.x == 0)
    {
        for (int i = 0; i < (1 << N) * (1 << k); i++)
        {
            atomicAdd(&partition_mass[i], block_partition_mass[i]);
        }
    }
}

int main()
{
    float *post_probs, *partition_mass;
    constexpr int prior_numer = 10;
    std::cout << "N = " << NUM << ", k = " << K << ", prior = " << (float)(prior_numer) / 100.0 << std::endl;
    int numElements = (1 << (NUM * K));
    hipError_t cudaStatus = hipMalloc((void **)&post_probs, numElements * sizeof(float));

    cudaStatus = hipMalloc((void **)&partition_mass, (1 << NUM) * (1 << K) * sizeof(float));
    cudaStatus = hipMemset(partition_mass, 0, (1 << NUM) * (1 << K) * sizeof(float));

    bin_enc *d_candidate;
    hipMalloc((void **)&d_candidate, sizeof(bin_enc));

    dim3 blockDims(BLOCK);                                          // Adjust block dimensions as needed
    dim3 gridDims((numElements + blockDims.x - 1) / blockDims.x); // Calculate grid dimensions
    static_assert(BLOCK <= (1 << SMEM), "Allocated shared memory too small!\n");

    std::chrono::time_point<std::chrono::system_clock> start, end;
    start = std::chrono::system_clock::now();
    set_prior_probs<NUM, K, prior_numer><<<gridDims, blockDims>>>(post_probs);

    end = std::chrono::system_clock::now();
    std::chrono::duration<float> elapsedSeconds = end - start;

    std::cout << "Prior kernel execution time: " << elapsedSeconds.count() << " seconds" << std::endl;

    start = std::chrono::system_clock::now();

    BBPA_smem_interleave<NUM, K, SMEM><<<gridDims, blockDims>>>(post_probs, partition_mass);

    hipDeviceSynchronize(); // Wait for the kernel to finish

    end = std::chrono::system_clock::now();
    elapsedSeconds = end - start;

    std::cout << "BBPA_smem_interleave time: " << elapsedSeconds.count() << " seconds" << std::endl;


    // start = std::chrono::system_clock::now();

    // BBPA_target<NUM, K><<<gridDims, blockDims>>>(post_probs, partition_mass);

    // cudaDeviceSynchronize(); // Wait for the kernel to finish

    // end = std::chrono::system_clock::now();
    // elapsedSeconds = end - start;

    // std::cout << "BBPA_target time: " << elapsedSeconds.count() << " seconds" << std::endl;


    // Copy the result back from the GPU
    bin_enc h_candidate;
    hipMemcpy(&h_candidate, d_candidate, sizeof(bin_enc), hipMemcpyDeviceToHost);

    float *h_partition_mass = new float[(1 << NUM) * (1 << K)];
    hipMemcpy(h_partition_mass, partition_mass, (1 << NUM) * (1 << K) * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < 40; i += 4)
    {
        std::cout << h_partition_mass[i] + h_partition_mass[i + 1] + h_partition_mass[i + 2] + h_partition_mass[i + 3] << "  ";
    }

    std::cout << "\nCandidate is: " << h_candidate << std::endl;

    // Free allocated memory on the GPU
    hipFree(post_probs);
    hipFree(partition_mass);
    hipFree(d_candidate);

    return 0;
}
