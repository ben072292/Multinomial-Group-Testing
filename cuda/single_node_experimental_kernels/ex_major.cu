#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>

#ifndef N
#define N 13
#endif

#ifndef K
#define K 2
#endif

#ifndef P
#define P 10
#endif

#ifndef B
#define B 8
#endif

typedef int bin_enc;

template <int n, int k, int p>
__global__ void set_prior_probs(float *_post_probs)
{
    const float pi0 = p / 100.0f;
    int s_iter = blockIdx.x * blockDim.x + threadIdx.x;
    float prob = 1.0f;
    for (int i = 0; i < n * k; i++)
    {
        if ((s_iter & (1 << i)) == 0)
            prob *= pi0;
        else
            prob *= (1.0f - pi0);
    }
    _post_probs[s_iter] = prob;
}

/** A100: N = 15, k = 2, prior = 0.1: 89.0362 seconds */
/** RTX3060: N=13, k = 2, prior = 0.1: 10.4249 seconds */
/** RTX3060: N = 15, k = 2, prior = 0.1: 562.238 seconds */
template <int n, int k>
__global__ void BBPA(const float *_post_probs, float *mins)
{
    int partition_id = 0;
    float partition_mass[1 << k];
    memset(partition_mass, 0, (1 << k) * sizeof(float));
    int ex = blockIdx.x * blockDim.x + threadIdx.x;
    for (int s_iter = 0; s_iter < (1 << (n * k)); s_iter++)
    {
        // #pragma unroll
        for (int variant = 0; variant < k; variant++)
        {
            partition_id |= ((1 << variant) & (((ex & (s_iter >> (variant * n))) - ex) >> 31));
        }

        // partition_id |= (1 & (((ex & s_iter) - ex) >> 31));
        // partition_id |= (2 & (((ex & (s_iter >> n)) - ex) >> 31));
        partition_mass[partition_id] += _post_probs[s_iter];
        partition_id = 0;
    }

    float min = 0.0;
    for(int i = 0; i < (1 << k); i++){
        min += abs(partition_mass[i] - 1.0 / (1 << k));
    }
    mins[ex] = min;
}

int main()
{
    float *post_probs, *mins;

    std::cout << "N = " << N << ", k = " << K << ", prior = " << (float)(P) / 100.0 << std::endl;
    int numElements = (1 << (N * K));
    hipError_t cudaStatus = hipMalloc((void **)&post_probs, numElements * sizeof(float));

    cudaStatus = hipMalloc((void **)&mins, (1 << K) * sizeof(float));
    // cudaStatus = cudaMemset(mins, 0, (1 << K) * sizeof(float));

    bin_enc *d_candidate;
    hipMalloc((void **)&d_candidate, sizeof(bin_enc));

    dim3 blockDims(B);                                          // Adjust block dimensions as needed
    dim3 gridDims((numElements + blockDims.x - 1) / blockDims.x); // Calculate grid dimensions

    std::chrono::time_point<std::chrono::system_clock> start, end;
    start = std::chrono::system_clock::now();
    set_prior_probs<N, K, P><<<gridDims, blockDims>>>(post_probs);

    end = std::chrono::system_clock::now();
    std::chrono::duration<float> elapsedSeconds = end - start;

    std::cout << "Prior kernel execution time: " << elapsedSeconds.count() << " seconds" << std::endl;

    numElements = (1 << N);
    dim3 gridDims1((numElements + blockDims.x - 1) / blockDims.x); // Calculate grid dimensions

    start = std::chrono::system_clock::now();

    BBPA<N, K><<<gridDims1, blockDims>>>(post_probs, mins);

    hipDeviceSynchronize(); // Wait for the kernel to finish

    end = std::chrono::system_clock::now();
    elapsedSeconds = end - start;

    std::cout << "BBPA kernel execution time: " << elapsedSeconds.count() << " seconds" << std::endl;

    // Copy the result back from the GPU
    bin_enc h_candidate;
    hipMemcpy(&h_candidate, d_candidate, sizeof(bin_enc), hipMemcpyDeviceToHost);

    float *h_mins = new float[(1 << N)];
    hipMemcpy(h_mins, mins, (1 << N) * sizeof(float), hipMemcpyDeviceToHost);
    float global_min = 2.0;
    float global_candidate = -1;
    for (int i = 0; i < (1 << N); i++)
    {
        if(h_mins[i] < global_min){
            global_min = h_mins[i];
            global_candidate = i;
        }
    }

    std::cout << "\nCandidate is: " << global_candidate << std::endl;

    // Free allocated memory on the GPU
    hipFree(post_probs);
    hipFree(mins);
    delete[] h_mins;

    return 0;
}
