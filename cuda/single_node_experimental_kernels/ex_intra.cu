#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>

#ifndef N
#define N 13
#endif

#ifndef K
#define K 2
#endif

#ifndef P
#define P 10
#endif

#ifndef B
#define B 8
#endif

typedef int bin_enc;

template <int n, int k, int p>
__global__ void set_prior_probs(float *_post_probs)
{
    const float pi0 = p / 100.0f;
    int s_iter = blockIdx.x * blockDim.x + threadIdx.x;
    float prob = 1.0f;
    for (int i = 0; i < n * k; i++)
    {
        if ((s_iter & (1 << i)) == 0)
            prob *= pi0;
        else
            prob *= (1.0f - pi0);
    }
    _post_probs[s_iter] = prob;
}

/** A100: N = 15, k = 2, prior = 0.1: 46.9054 seconds
 *  RTX3060: N = 15, k = 2, prior = 0.1: 154.283 seconds
 */
template <int n, int k, int b>
__global__ void BBPA(const float *probs, float *mass)
{
    int laneId = threadIdx.x & 0x1f;
    float r_mass[1 << k];
    memset(r_mass, 0, (1 << k) * sizeof(float));
    int ex = blockIdx.x;
    for (int s_iter = 0; s_iter < (1 << (n * k - b)); s_iter++)
    {
        int state = threadIdx.x * (1 << (n * k - b)) + s_iter;
        int partition_id = 0;
        #pragma unroll k
        for (int variant = 0; variant < k; variant++)
        {
            partition_id |= ((1 << variant) & (((ex & (state >> (variant * n))) - ex) >> 31));
        }
        // partition_id |= (1 & (((ex & state) - ex) >> 31));
        // partition_id |= (2 & (((ex & (state >> n)) - ex) >> 31));
        r_mass[partition_id] += probs[state];
    }

    for (int i = 0; i < (1 << k); i++)
    {
        for (int j = 16; j >= 1; j /= 2)
        {
            r_mass[i] += __shfl_xor_sync(0xffffffff, r_mass[i], j, 32);
        }
    }
    if(!laneId){
        atomicAdd(&mass[blockIdx.x], r_mass[0] + r_mass[1] + r_mass[2] + r_mass[3]);
    }
}

int main()
{
    float *d_probs, *d_mass;

    std::cout << "N = " << N << ", k = " << K << ", prior = " << P / 100.0f << std::endl;
    int numElements = (1 << (N * K));
    hipError_t cudaStatus = hipMalloc((void **)&d_probs, numElements * sizeof(float));

    cudaStatus = hipMalloc((void **)&d_mass, (1 << (N + K)) * sizeof(float));
    // cudaStatus = cudaMemset(mins, 0, (1 << curr_subjs) * sizeof(float));

    bin_enc *d_candidate;
    hipMalloc((void **)&d_candidate, sizeof(bin_enc));

    dim3 blockDims(1 << B);                                       // Adjust block dimensions as needed
    dim3 gridDims((numElements + blockDims.x - 1) / blockDims.x); // Calculate grid dimensions

    std::chrono::time_point<std::chrono::system_clock> start, end;
    start = std::chrono::system_clock::now();
    set_prior_probs<N, K, P><<<gridDims, blockDims>>>(d_probs);

    end = std::chrono::system_clock::now();
    std::chrono::duration<float> elapsedSeconds = end - start;

    std::cout << "Prior kernel execution time: " << elapsedSeconds.count() << " seconds" << std::endl;

    dim3 gridDims1(1 << N); // Calculate grid dimensions

    start = std::chrono::system_clock::now();

    BBPA<N, K, B><<<gridDims1, blockDims>>>(d_probs, d_mass);

    hipDeviceSynchronize(); // Wait for the kernel to finish

    end = std::chrono::system_clock::now();
    elapsedSeconds = end - start;

    std::cout << "BBPA kernel execution time: " << elapsedSeconds.count() << " seconds" << std::endl;

    // Copy the result back from the GPU
    float *h_partition_mass = new float[1 << (N + K)];
    hipMemcpy(h_partition_mass, d_mass, (1 << (N + K)) * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; i ++)
    {
        std::cout << h_partition_mass[i] << "  ";
    }
    std::cout << std::endl;

    // Free allocated memory on the GPU
    hipFree(d_probs);
    hipFree(d_mass);

    return 0;
}
