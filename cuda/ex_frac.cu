#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "mpi.h"
#include "nccl.h"
#include <chrono>
#include <iostream>
#include <stdint.h>
#include <unistd.h>

#ifndef N
#define N 12
#endif

#ifndef K
#define K 2
#endif

#ifndef P
#define P 10
#endif

#ifndef B
#define B 256
#endif

#ifndef F
#define F 10
#endif

#define MPICHECK(cmd)                                \
    do                                               \
    {                                                \
        int e = cmd;                                 \
        if (e != MPI_SUCCESS)                        \
        {                                            \
            printf("Failed: MPI error %s:%d '%d'\n", \
                   __FILE__, __LINE__, e);           \
            exit(EXIT_FAILURE);                      \
        }                                            \
    } while (0)

#define CUDACHECK(cmd)                                         \
    do                                                         \
    {                                                          \
        hipError_t e = cmd;                                   \
        if (e != hipSuccess)                                  \
        {                                                      \
            printf("Failed: Cuda error %s:%d '%s'\n",          \
                   __FILE__, __LINE__, hipGetErrorString(e)); \
            exit(EXIT_FAILURE);                                \
        }                                                      \
    } while (0)

#define NCCLCHECK(cmd)                                         \
    do                                                         \
    {                                                          \
        ncclResult_t r = cmd;                                  \
        if (r != ncclSuccess)                                  \
        {                                                      \
            printf("Failed, NCCL error %s:%d '%s'\n",          \
                   __FILE__, __LINE__, ncclGetErrorString(r)); \
            exit(EXIT_FAILURE);                                \
        }                                                      \
    } while (0)

static uint64_t getHostHash(const char *string)
{
    // Based on DJB2a, result = result * 33 ^ char
    uint64_t result = 5381;
    for (int c = 0; string[c] != '\0'; c++)
    {
        result = ((result << 5) + result) ^ string[c];
    }
    return result;
}

static void getHostName(char *hostname, int maxlen)
{
    gethostname(hostname, maxlen);
    for (int i = 0; i < maxlen; i++)
    {
        if (hostname[i] == '.')
        {
            hostname[i] = '\0';
            return;
        }
    }
}

typedef int bin_enc;

__device__ bin_enc offset_to_state(int offset, int rank, int nranks){
    return (1 << (N * K)) * rank / nranks + offset;
}

template <int n, int k, int p>
__global__ void set_prior_probs(float *_post_probs, int rank, int nranks)
{
    const float pi0[30] = {0.01f, 0.02f, 0.03f, 0.04f, 0.05f, 0.06f, 0.07f, 0.08f, 0.09f, 0.1f,
                           0.11f, 0.12f, 0.13f, 0.14f, 0.15f, 0.16f, 0.17f, 0.18f, 0.19f, 0.2f,
                           0.21f, 0.22f, 0.23f, 0.24f, 0.25f, 0.26f, 0.27f, 0.28f, 0.29f, 0.3f};
    int s_iter = blockIdx.x * blockDim.x + threadIdx.x;
    float prob = 1.0f;
    for (int i = 0; i < n * k; i++)
    {
        if ((offset_to_state(s_iter, rank, nranks) & (1 << i)) == 0)
            prob *= pi0[i];
        else
            prob *= (1.0f - pi0[i]);
    }
    _post_probs[s_iter] = prob;
}

/** A100: N = 15, k = 2, prior = 0.1: 46.9054 seconds
 *  RTX3060: N = 15, k = 2, prior = 0.1: 154.283 seconds
 */
template <int n, int k, int f>
__global__ void halving(const float *probs, float *mass, int rank, int nranks)
{
    float r_mass[1 << k];
    memset(r_mass, 0, (1 << k) * sizeof(float));
    int ex = (blockIdx.x * blockDim.x + threadIdx.x) % (1 << n);
    int iter = (blockIdx.x * blockDim.x + threadIdx.x) / (1 << n);
    int iters = (1 << (n * k - f)) / nranks;
    for (int s_iter = 0; s_iter < iters; s_iter++)
    {
        int state = iter * iters + s_iter;
        int partition_id = 0;
#pragma unroll k
        for (int variant = 0; variant < k; variant++)
        {
            partition_id |= ((1 << variant) & (((ex & (offset_to_state(state, rank, nranks) >> (variant * n))) - ex) >> 31));
        }
        // partition_id |= (1 & (((ex & state) - ex) >> 31));
        // partition_id |= (2 & (((ex & (state >> n)) - ex) >> 31));
        r_mass[partition_id] += probs[state];
    }

    // atomicAdd(reinterpret_cast<float4*>(mass + ex * sizeof(float4)), *reinterpret_cast<float4*>(r_mass)); // only supported starting compute capability 9.0
    for (int i = 0; i < (1 << k); i++)
    {
        atomicAdd(&mass[ex * (1 << k) + i], r_mass[i]);
        // mass[ex * (1 << k) + i] += r_mass[i];
    }
}

int main(int argc, char *argv[])
{
    int myRank, nRanks, localRank = 0;

    // initializing MPI
    MPICHECK(MPI_Init(&argc, &argv));
    MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
    MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));

    // calculating localRank based on hostname which is used in selecting a GPU
    uint64_t hostHashs[nRanks];
    char hostname[1024];
    getHostName(hostname, 1024);
    hostHashs[myRank] = getHostHash(hostname);
    MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
    for (int p = 0; p < nRanks; p++)
    {
        if (p == myRank)
            break;
        if (hostHashs[p] == hostHashs[myRank])
            localRank++;
    }

    // std::cout << "Rank " << myRank << "->GPU " << localRank << std::endl;

    ncclUniqueId id;
    ncclComm_t comm;
    hipStream_t s;
    float *d_probs, *d_mass;
    bin_enc *d_candidate;
    int numElements = (1 << (N * K)) / nRanks;
    hipMalloc((void **)&d_candidate, sizeof(bin_enc));
    dim3 blockDims(B);                                          // Adjust block dimensions as needed
    dim3 gridDims((numElements + blockDims.x - 1) / blockDims.x); // Calculate grid dimensions

    // get NCCL unique ID at rank 0 and broadcast it to all others
    if (myRank == 0)
        ncclGetUniqueId(&id);
    MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));

    // picking a GPU based on localRank, allocate device buffers
    CUDACHECK(hipSetDevice(localRank));
    CUDACHECK(hipMalloc((void **)&d_probs, (1 << (N * K)) * sizeof(float) / nRanks));
    CUDACHECK(hipMalloc((void **)&d_mass, (1 << (N + K)) * sizeof(float)));
    // CUDACHECK(hipMemset(mins, 0, (1 << curr_subjs) * sizeof(float)));
    CUDACHECK(hipStreamCreate(&s));

    // initializing NCCL
    NCCLCHECK(ncclCommInitRank(&comm, nRanks, id, myRank));

    if (!myRank)
    {
        std::cout << "N = " << N << ", k = " << K << ", prior = " << P / 100.0f << std::endl;
        std::cout << "Number of GPUs: " << nRanks << std::endl;
    }

    std::chrono::time_point<std::chrono::system_clock> start, end_1, end_2, end_3;
    start = std::chrono::system_clock::now();

    set_prior_probs<N, K, P><<<gridDims, blockDims, 0, s>>>(d_probs, myRank, nRanks);

    CUDACHECK(hipStreamSynchronize(s));

    end_1 = std::chrono::system_clock::now();
    std::chrono::duration<float> elapsedSeconds = end_1 - start;
    if (!myRank)
        std::cout << "Prior kernel execution time: " << elapsedSeconds.count() << " seconds" << std::endl;

    numElements = (1 << (N + F));
    dim3 gridDims1((numElements + blockDims.x - 1) / blockDims.x); // Calculate grid dimensions

    halving<N, K, F><<<gridDims1, blockDims, 0, s>>>(d_probs, d_mass, myRank, nRanks);

    CUDACHECK(hipStreamSynchronize(s));

    end_2 = std::chrono::system_clock::now();
    elapsedSeconds = end_2 - end_1;
    if (!myRank)
        std::cout << "BBPA kernel execution time: " << elapsedSeconds.count() << " seconds" << std::endl;

    // communicating using NCCL
    NCCLCHECK(ncclAllReduce((const void *)d_mass, (void *)d_mass, (1 << (N + K)), ncclFloat, ncclSum,
                            comm, s));

    CUDACHECK(hipStreamSynchronize(s));

    end_3 = std::chrono::system_clock::now();
    elapsedSeconds = end_3 - end_2;
    if (!myRank)
        std::cout << "BBPA kernel Allreduce time: " << elapsedSeconds.count() << " seconds" << std::endl;

    elapsedSeconds = end_3 - end_1;
    if (!myRank)
        std::cout << "BBPA kernel total execution time: " << elapsedSeconds.count() << " seconds" << std::endl;
    
    // Copy the result back from the GPU
    float *h_partition_mass = new float[10 * (1 << K)];
    CUDACHECK(hipMemcpy(h_partition_mass, d_mass, (1 << K) * sizeof(float) * 10, hipMemcpyDeviceToHost));

    if (!myRank)
    {
        for (int i = 0; i < 10 * (1 << K); i += (1 << K))
        {
            float total = 0.0;
            for(int j = i; j < i + (1 << K); j++){
                total += h_partition_mass[j];
            }
            std::cout << total << " ";
        }
        std::cout << std::endl;
    }

    // Free allocated memory on the GPU
    CUDACHECK(hipFree(d_probs));
    CUDACHECK(hipFree(d_mass));

    // Free Host
    delete[] h_partition_mass;

    // finalizing NCCL
    ncclCommDestroy(comm);

    // finalizing MPI
    MPICHECK(MPI_Finalize());

    return 0;
}
